#include "hip/hip_runtime.h"
__global__ void reduce5(float *d_ivec, float *d_ovec, unsigned int n) {
  extern __shared__ float sdata[];
  unsigned int tid = threadIdx.x;
  // each thread is responsible for positions in four blocks
  // eg. blk0.thread0: blk0[0] and blk1[0],
  //     blk1.thread0: blk2[0] and blk3[0]
  //     etc.
  unsigned int i   = blockIdx.x *
  	             (blockDim.x*4)   // note * 4 
		     + threadIdx.x;
  
  // don't just copy but accumulate values from global mem into shared memory
  // as long as there are blocks after ours
  sdata[tid] = ((i<n) ? d_ivec[i] : 0.0) +
               ((i+blockDim.x < n) ? d_ivec[i+blockDim.x] : 0.0) +
	       ((i+(2*blockDim.x) < n) ? d_ivec[i+(2*blockDim.x)] : 0.0) +
       	       ((i+(3*blockDim.x) < n) ? d_ivec[i+(3*blockDim.x)] : 0.0);			
  
  __syncthreads();

  BPRINT("%d.%d sdata coppied\n",blockIdx.x, tid);

  for (unsigned int s=blockDim.x/2; s>0; s >>= 1) {
    if (tid < s) {  
      sdata[tid] += sdata[tid+s];
    }
    __syncthreads();    
  }
  
  BPRINT("%d.%d sdata computed: sdata[0]=%f &d_ovec[%d]=%p\n", blockIdx.x, tid, sdata[0], blockIdx.x,  &(d_ovec[blockIdx.x]));
  if (tid==0) d_ovec[blockIdx.x] = sdata[0];
}

void reduce5bytes(uint64_t *readbytes, uint64_t *writtenbytes, int n, int blksize)
{ 
  uint64_t rb = 0, wb = 0;
  for (int len=n; len>1; len=len/blksize) {
    rb += len;                   // reads all len elements 
    if (n/blksize <= 1) wb += 1; // one for one block or partial block
    else  wb += len/(4*blksize); // one for every two blocks
    len = len / 4;               // each step takes care of 4 blocks worth
  }
  rb *= sizeof(float); wb *= sizeof(float);
  *readbytes = rb; *writtenbytes = wb;
}
